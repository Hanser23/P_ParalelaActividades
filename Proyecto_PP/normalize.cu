#include <hip/hip_runtime.h>
__global__ void normalize(float* x, float minv, float maxv, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n){
    float v = x[i];
    x[i] = (v - minv) / (maxv - minv + 1e-8f);
  }
}
extern "C" void normalize_cuda(float* d_x, float minv, float maxv, int n){
  int blk = 256;
  int grid = (n + blk - 1) / blk;
  normalize<<<grid, blk>>>(d_x, minv, maxv, n);
  hipDeviceSynchronize();
}
